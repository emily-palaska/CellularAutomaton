
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define bSize 256 // block size for easy access

void takeInput(const char *filename, int *array, int n);
void printArray(int *array, long n);

// CUDA kernel to perform vector ising step
__global__ void isingStep(int *dArrayInput, int *dArrayOutput, long n) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within the array bounds
    if (idx < n * n) {
    	int buffer;
		int __shared__ blockMemory[bSize];
    	int neighbors[5];
    	
    	// copy host data to shared memory
    	blockMemory[idx % bSize] = dArrayInput[idx];
    	__syncthreads();
    	
    	long i = idx % n;
    	long j = idx / n;
    	long up, down, left, right;
    	
    	left = (i - 1 + n) % n + j * n;		
		right = (i + 1) % n + j * n;			
		up = i + ((j - 1 + n) % n) * n;
		down = i + ((j + 1) % n) * n;
    	
    	// check if host memory needs to be accessed
    	if (left % bSize == idx % bSize)
    		neighbors[0] = blockMemory[left % bSize];
    	else
    		neighbors[0] = dArrayInput[left];
    	
    	if (right % bSize == idx % bSize)
    		neighbors[1] = blockMemory[right % bSize];
    	else
    		neighbors[1] = dArrayInput[right];
    	
    	neighbors[2] = blockMemory[idx % bSize];
    	
    	if (up % bSize == idx % bSize)
    		neighbors[3] = blockMemory[up % bSize];
    	else
    		neighbors[3] = dArrayInput[up];
    		
    	if (down % bSize == idx % bSize)
    		neighbors[4] = blockMemory[down % bSize];
    	else
    		neighbors[4] = dArrayInput[down];
    		
    	__syncthreads();
    	
    	// add all neighboring values
        buffer = 0;
		for(int k = 0; k < 5; k++)
			buffer += neighbors[k];
		
		// keep the sign
		dArrayOutput[idx] = buffer / abs(buffer);
    }
}

int main(int argc, char *argv[]) {
	if (argc != 2) {
		printf("Invalid arguments\n");
		return 1;
	}
	
	const char *filename = "input.txt"; //input file name
	long n = atoi(argv[1]); // dimension
	int k = 500; // number of steps
	int counter = 0;
	struct timeval t1, t2;
		
	// MEMORY ALLOCATION
	
	// host array initialization
	int *hArray = (int *)malloc(n * n * sizeof(int));
	
	if(!hArray) {
		perror("Memory allocation failed");
		return 1;
	}	
		
	// device array initialization	
	int *dArray1;
	int *dArray2;
	
	hipMalloc((void **)&dArray1, n * n * sizeof(int));
	hipMalloc((void **)&dArray2, n * n * sizeof(int));
	
	// INITIAL STATE
	takeInput(filename, hArray, n);
	//printf("Initial state:\n");
	//printArray(hArray, n);
	//printf("\n\nFinal state:\n");
	
	gettimeofday(&t1, NULL);
	// ALGORITHM APPLICATION
	
	// copy host vectors to device
    hipMemcpy(dArray1, hArray, n * n * sizeof(int), hipMemcpyHostToDevice);
	
	// Define the grid and block dimensions
    int blockSize = bSize;
    int gridSize = (n * n + blockSize - 1) / blockSize;
    
    while(counter < k) {
		// Launch the kernel
    	isingStep<<<gridSize, blockSize>>>(dArray1, dArray2, n);
    	counter++;
    	if(counter == k) break;
    	isingStep<<<gridSize, blockSize>>>(dArray2, dArray1, n);
    	counter++;
	}
    // Copy the result back to the host
    if(counter % 2)    
    	hipMemcpy(hArray, dArray2, n * n * sizeof(int), hipMemcpyDeviceToHost);
    else
    	hipMemcpy(hArray, dArray1, n * n * sizeof(int), hipMemcpyDeviceToHost);
	
	gettimeofday(&t2, NULL);
	double elapsedTime;
	elapsedTime = (t2.tv_sec - t1.tv_sec);      // sec
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000000.0;   // us to sec
	
	// RESULT PRINTING
	//printf("\nResult:\n");
	//printArray(hArray, n);
	printf("Successful temination for: n = %d k = %d\nTime elapsed: %.4f seconds\n", n, k, elapsedTime);
	
	// MEMORY DE-ALLOCATION
	
	// Host
	free(hArray);

	// Device
    hipFree(dArray1);
    hipFree(dArray2);
	return 0;
}

void takeInput(const char *filename, int *array, int n) {
	FILE *file = fopen(filename, "r");
	if(file == NULL) {
		perror("Error opening file");
		exit(EXIT_FAILURE);
	}
	
	for(int i = 0; i < n * n; i ++)
		if(fscanf(file, "%d", &array[i]) != 1) {
				fprintf(stderr, "Error reading from file");
				exit(EXIT_FAILURE);
		}
	fclose(file);
	return;
}//1757156-



void printArray(int *array, long n) {
	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
				if(array[i * n + j] >= 0) printf("+");
				printf("%d ", array[i * n + j]);
			}
		printf("\n");
	}
	return;
}

