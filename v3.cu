
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define bSize 1024 // block size for easy access

void takeInput(const char *filename, int *array, long n);
void printArray(int *array, long n);

// CUDA kernel to perform vector ising step
__global__ void isingStep(int *dArrayInput, int *dArrayOutput, long n) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within the array bounds
    if (idx < n * n) {
    	int buffer;
		int __shared__ blockMemory[bSize];
    	int neighbors[5];
    	
    	// copy host data to shared memory
    	blockMemory[idx % bSize] = dArrayInput[idx];
    	__syncthreads();
    	
    	// auxiliary indices for checking border values
    	long i = idx % n;
    	long j = idx / n;
    	long up, down, left, right;
    	
    	left = (i - 1 + n) % n + j * n;		
		right = (i + 1) % n + j * n;			
		up = i + ((j - 1 + n) % n) * n;
		down = i + ((j + 1) % n) * n;
    	
    	// when in the same block, shared memory is acessed
    	// else cpu memory is accesed for border values
    	if (left / bSize == idx / bSize)
    		neighbors[0] = blockMemory[left % bSize];
    	else
    		neighbors[0] = dArrayInput[left];
    	
    	if (right / bSize == idx / bSize)
    		neighbors[1] = blockMemory[right % bSize];
    	else
    		neighbors[1] = dArrayInput[right];
    	
    	neighbors[2] = blockMemory[idx % bSize];
    	
    	if (up / bSize == idx / bSize)
    		neighbors[3] = blockMemory[up % bSize];
    	else
    		neighbors[3] = dArrayInput[up];
    		
    	if (down / bSize == idx / bSize)
    		neighbors[4] = blockMemory[down % bSize];
    	else
    		neighbors[4] = dArrayInput[down];
    		
    	__syncthreads();
    	
    	// add all neighboring values
        buffer = 0;
		for(int k = 0; k < 5; k++)
			buffer += neighbors[k];
		
		// keep the sign
		dArrayOutput[idx] = buffer / abs(buffer);
    }
}

int main(int argc, char *argv[]) {
	if (argc != 2) {
		printf("Invalid arguments\n");
		return 1;
	}
	
	const char *filename = "input.txt"; //input file name
	long n = atoi(argv[1]); 			// dimension taken as argument
	int k = 500; 						// number of steps
	int counter = 0;
	struct timeval t1, t2;				// variables for elapsed time
	
	///////////////////////	
	// MEMORY ALLOCATION //
	///////////////////////
	
	// host
	int *hArray = (int *)malloc(n * n * sizeof(int));
	if(!hArray) {
		perror("Memory allocation failed");
		return 1;
	}
			
	// device	
	int *dArray1;
	int *dArray2;	
	hipMalloc((void **)&dArray1, n * n * sizeof(int));
	hipMalloc((void **)&dArray2, n * n * sizeof(int));
	
	///////////////////
	// INITIAL STATE //
	///////////////////
	
	takeInput(filename, hArray, n);
	//printf("Initial state:\n");
	//printArray(hArray, n);
	//printf("\n\nFinal state:\n");
	
	///////////////
	// ALGORITHM //
	///////////////
	
	// start timer
	gettimeofday(&t1, NULL);
	
	// copy host vectors to device
    hipMemcpy(dArray1, hArray, n * n * sizeof(int), hipMemcpyHostToDevice);
	
	// define the grid and block dimensions
    int blockSize = bSize;
    int gridSize = (n * n + blockSize - 1) / blockSize;
    
    while(counter < k) {
		// Launch the kernel
    	isingStep<<<gridSize, blockSize>>>(dArray1, dArray2, n);
    	counter++;
    	if(counter == k) break;
    	isingStep<<<gridSize, blockSize>>>(dArray2, dArray1, n);
    	counter++;
	}
    // copy the result back to the host
    if(counter % 2)    
    	hipMemcpy(hArray, dArray2, n * n * sizeof(int), hipMemcpyDeviceToHost);
    else
    	hipMemcpy(hArray, dArray1, n * n * sizeof(int), hipMemcpyDeviceToHost);
	
	// stop timer
	gettimeofday(&t2, NULL);
	double elapsedTime;
	elapsedTime = (t2.tv_sec - t1.tv_sec);      			// sec
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000000.0;   // us to sec
	
	////////////////
	// DISPLAYING //
	////////////////
	
	//printArray(hArray, n);
	printf("Successful temination for: n = %d k = %d\nTime elapsed: %.4f seconds\n", n, k, elapsedTime);
	
	//////////////////////////
	// MEMORY DE-ALLOCATION //
	//////////////////////////
	
	free(hArray);		// host
    hipFree(dArray1);	// device
    hipFree(dArray2);
	return 0;
}

// function that takes the initial values of the grid from a txt file
void takeInput(const char *filename, int *array, long n) {
	FILE *file = fopen(filename, "r");
	if(file == NULL) {
		perror("Error opening file");
		exit(EXIT_FAILURE);
	}
	
	for(int i = 0; i < n * n; i ++)
		if(fscanf(file, "%d", &array[i]) != 1) {
				fprintf(stderr, "Error reading from file");
				exit(EXIT_FAILURE);
		}
	fclose(file);
	return;
}

// auxialiary function to print the grid - mainly for checks
void printArray(int *array, long n) {
	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
				if(array[i * n + j] >= 0) printf("+");
				printf("%d ", array[i * n + j]);
			}
		printf("\n");
	}
	return;
}

